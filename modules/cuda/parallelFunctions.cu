#include "hip/hip_runtime.h"
/**
* parallelizer.cu
* Guardian Kids
* NestNet Group
* parallelization functions to speed up algorithms
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "../../include/cuda/hip/hip_runtime_api.h"

#include "../../include/dataset.h"
#include "../../include/haarFeature.h"
#include "../../include/haarFeatureGenerator.h"

extern "C" {
#include "../../include/cuda/parallelFunctions.h"
#include "../../include/cuda/mergeSort_common.h"
}

// the number of samples must be multiple of this value
#define MAX_THREADS_PER_BLOCK 32

// variables for debugging
#define FEATURE 10818
#define LOWERID 5543
#define UPPERID 5544
#define BLOCKID 173

// number of samples
unsigned int N;

// dimension of images
unsigned int D;

// the features
HaarFeature *mFeatures;

// device variables
__managed__ HaarFeature *d_f;
float *d_X;
unsigned char *d_Y;
unsigned int *d_i;
float *d_g;
float *d_w;
float *d_s1;
float *d_s0;
float *d_ss1;
float *d_ss0;
float *d_e;
char *d_p;
unsigned int *d_bei;
float *d_BufKey;
unsigned int *d_BufVal;

// host variables
float *h_bs1;
float *h_bs0;
float *h_g;
float *h_errors;
unsigned int *h_bei;
char *h_p;

bool flag = false;
unsigned int blocksPerGrid;
unsigned int fIndex;
unsigned int tIndex;
unsigned int sizef;
unsigned int sizef2;
unsigned int sizei;
unsigned int sizeui;
unsigned int sizeui2;
unsigned int sizeuc;
unsigned int sizec;

__global__ void computeFeatureValuesKernel(
	float *X, 
	float *d_g, 
	unsigned int *d_i, 
	unsigned int n, 
	unsigned int dim, 
	unsigned int featureIndex)
{
	unsigned long int idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(idx < n)
	{
		// initialize the indexes
		d_i[idx] = idx;
		
		float intensityWhite = 0.0;
		float intensityBlack = 0.0;
		
		long unsigned int stride = dim * dim * idx;
		unsigned int count = d_f[featureIndex].numberOfRects;
	
		for(unsigned int i = 0; i < count; i++)
		{
			HaarRectangle r = d_f[featureIndex].rectangles[i];
			
			float d = 0.0, a = 0.0, b = 0.0, c = 0.0;

			if(r.x - 1 >= 0 && r.y - 1 >= 0)
				d = X[stride + (dim * (r.y - 1)) + (r.x - 1)];
			
			if(r.y - 1 >= 0)
				b = X[stride + (dim * (r.y - 1)) + (r.x + r.width - 1)];
			
			if(r.x - 1 >= 0)
				c = X[stride + (dim * (r.y + r.height - 1)) + (r.x - 1)];
	
			a = X[stride + (dim * (r.y + r.height - 1)) + (r.x + r.width - 1)];
		
			if(r.isBlack)
				intensityBlack += (a - b - c + d);
			else
				intensityWhite += (a - b - c + d);
		}
		
		// save the feature value
		d_g[idx] = (intensityWhite - intensityBlack);
	}
	
	//__syncthreads();
}

__global__ void computeWeightScanKernel(
	unsigned char *d_Y, 
	float *d_w, 
	unsigned int *d_i, 
	float *d_s1,
	float *d_s0,
	float *d_ss1,
	float *d_ss0,
	unsigned int blocks, 
	unsigned int fi)
{
	__shared__ float s1[MAX_THREADS_PER_BLOCK];
	__shared__ float s0[MAX_THREADS_PER_BLOCK];
	
	unsigned int tid = threadIdx.x;
	unsigned long int idx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int sIndex = d_i[idx];
	
	if(d_Y[sIndex] == 1)
	{
		s1[tid] = d_w[sIndex]; 
		s0[tid] = 0.0;
	}
	
	if(d_Y[sIndex] == 0)
	{
		s0[tid] = d_w[sIndex];
		s1[tid] = 0.0;
	}
	__syncthreads();
	
	//if(fi == FEATURE && idx >= LOWERID && idx < UPPERID) printf("block: %d, tid: %d, sIndex: %d, Y: %d, W: %.12f\n", blockIdx.x, tid, sIndex, d_Y[sIndex], d_w[sIndex]);
	// perform the first iteration to sum values in s1 and s0
	if(tid % 2 == 1)
	{
		s1[tid] = s1[tid] + s1[tid - 1];
		s0[tid] = s0[tid] + s0[tid - 1];
	}
	
	__syncthreads();
	
	// perform the rest of iterations to sum values in s1 and s0
	for(int s = 2; s < blockDim.x; s <<= 1)
	{
		if(tid < s)
			break;
		
		for(int i = s, j = 0, m = s - 1; i < blockDim.x; i++, j++)
		{
			if(tid < i)
				break;
				
			if(tid == i)
			{
				s1[tid] = s1[tid] + s1[m];
				s0[tid] = s0[tid] + s0[m];
			}
			
			if(j == s - 1)
			{
				i += s;
				m += s + s;
				j = -1;
			}
		}
		
		__syncthreads();
	}
	
	__syncthreads();
	
	// initialize the final scan value of the first block to zero, that means the first block of s1 and s0 is ready
	if(blockIdx.x == 0 && tid == blockDim.x - 1)
	{
		d_s1[blockIdx.x] = 0.0;
		d_s0[blockIdx.x] = 0.0;
	}
	
	// save the rest of the last scan value of every block into the global memory
	if(tid == blockDim.x - 1)
	{
		d_s1[blockIdx.x + 1] = s1[tid];
		d_s0[blockIdx.x + 1] = s0[tid];
	}
	
	__syncthreads();
	
	d_ss1[idx] = s1[tid];
	d_ss0[idx] = s0[tid];
	
	__syncthreads();
	
	//f(fi == FEATURE && idx >= LOWERID && idx < UPPERID) printf("(GPU) tid: %d, i: %d,  s1: %.12f, s0: %.12f\n", tid, sIndex, d_ss1[idx], d_ss0[idx]);
}

__global__ void computeErrorKernel(
	float *d_e, 
	unsigned int *d_bei, 
	float *d_s1,
	float *d_s0,
	float *d_ss1,
	float *d_ss0, 
	char *d_p, 
	float t1, 
	float t0, 
	unsigned int fi)
{
	__shared__ float e[MAX_THREADS_PER_BLOCK];
	__shared__ int bei[MAX_THREADS_PER_BLOCK];

	unsigned int tid = threadIdx.x;
	unsigned long int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	// increment the value given by every block to complete the full scan
	d_ss1[idx] = d_ss1[idx] + d_s1[blockIdx.x];
	d_ss0[idx] = d_ss0[idx] + d_s0[blockIdx.x];
	__syncthreads();

	// compute all possible errors using every posible threshold
	// by consider first, positive samples below the threshold and
	// secondly, negative samples below the threshold 
	float posBelowNegAbove = (d_ss0[idx] + (t1 - d_ss1[idx]));
	float negBelowPosAbove = (d_ss1[idx] + (t0 - d_ss0[idx]));

	if(posBelowNegAbove < negBelowPosAbove)
	{
		d_e[idx] = posBelowNegAbove;
		e[tid] = posBelowNegAbove;
		d_p[idx] = 1;
	}
	else
	{
		d_e[idx] = negBelowPosAbove;
		e[tid] = negBelowPosAbove;
		d_p[idx] = -1;
	}
	
	__syncthreads();
	
	// first iteration to get the index of the lower error
	for(int i = 1; i < blockDim.x; i += 2)
	{
		if(tid == i)
		{
			if(e[tid] > e[tid - 1])
			{
				bei[tid] = tid - 1;
			}
			else
			{
				bei[tid] = tid;
			}
			
			bei[tid - 1] = 0;
		}
	}
	
	__syncthreads();
	
	// subsequent iterations to get the index of the lower error
	for(int s = 4; s <= blockDim.x; s <<= 1)
	{
		for(int i = s - 1; i < blockDim.x; i += s)
		{
			if(tid == i)
			{
				int shift = s / 2;
				
				if(e[bei[tid]] > e[bei[tid - shift]])
				{
					bei[tid] = bei[tid - shift];
				}
				else
				{
					bei[tid] = bei[tid];
				}
				
				bei[tid - shift] = 0;
			}
		}
		
		__syncthreads();
	}
	
	__syncthreads();
	
	if(tid == blockDim.x - 1)
	{
		d_bei[blockIdx.x] = bei[tid];
		//if(fi == FEATURE && blockIdx.x == BLOCKID) printf("blockId: %d, bei[%d]: %d, error: %.12f\n", blockIdx.x, tid, bei[tid], e[bei[tid]]);
	}
	
	__syncthreads();
}

extern "C"
void initParallelization(float *X, UBYTE *Y, unsigned int n, unsigned int d)
{
	N = n;
	D = d;
	
	if(!flag)
		printf("allocating memory in device\n");
	
	checkCudaErrors(hipDeviceReset());
	
	sizef = N * sizeof(float);
	sizef2 = (N / MAX_THREADS_PER_BLOCK) * sizeof(float);
	sizei = N * sizeof(int);
	sizeui = N * sizeof(unsigned int);
	sizeui2 = (N / MAX_THREADS_PER_BLOCK) * sizeof(unsigned int);
	sizeuc = N * sizeof(UBYTE);
	sizec = N * sizeof(char);
	
	// get generated features
	getFeatures(&mFeatures);

	// allocate features in device memory
	int sizeF = sizeof(HaarFeature) * getFeaturesLength();
	checkCudaErrors(hipMallocManaged(&d_f, sizeF));

	for(unsigned int i = 0; i < getFeaturesLength(); i++)
	{
		d_f[i].numberOfRects = mFeatures[i].numberOfRects;
		int sizeR = sizeof(HaarRectangle) * mFeatures[i].numberOfRects;
		checkCudaErrors(hipMalloc((void **)&d_f[i].rectangles, sizeR));
		checkCudaErrors(hipMemcpy(d_f[i].rectangles, mFeatures[i].rectangles, sizeR, hipMemcpyHostToDevice));
	}

	printf("Allocated memory for features.\n");
	//getchar();
	// allocate image data in device memory
	unsigned int sizeX = N * D * D * sizeof(float);
	checkCudaErrors(hipMalloc((void **)&d_X, sizeX));
	checkCudaErrors(hipMemcpy(d_X, X, sizeX, hipMemcpyHostToDevice));
	printf("Allocated memory for images data.\n"); 
	//getchar();
	
	// allocate feature values and indexes
	checkCudaErrors(hipMalloc((void **)&d_g, sizef));
	checkCudaErrors(hipMalloc((void **)&d_i, sizeui));
	
	// allocate pointers for merge sorting
    checkCudaErrors(hipMalloc((void **)&d_BufKey, sizef));
    checkCudaErrors(hipMalloc((void **)&d_BufVal, sizeui));

	// allocate Y, W, S1, S0, e, neccesary for error computing 
	checkCudaErrors(hipMalloc((void **)&d_Y, sizeuc));
	checkCudaErrors(hipMalloc((void **)&d_w, sizef));
	checkCudaErrors(hipMalloc((void **)&d_s1, sizef2));
	checkCudaErrors(hipMalloc((void **)&d_s0, sizef2));
	checkCudaErrors(hipMalloc((void **)&d_e, sizef));
	checkCudaErrors(hipMalloc((void **)&d_p, sizec));
	checkCudaErrors(hipMalloc((void **)&d_bei, sizeui2));
	checkCudaErrors(hipMalloc((void **)&d_ss1, sizef));
	checkCudaErrors(hipMalloc((void **)&d_ss0, sizef));

	checkCudaErrors(hipMemcpy(d_Y, Y, sizeuc, hipMemcpyHostToDevice));
	
	// variables to hold the last value of the first scan in s1 and s0
	h_bs1 = (float *)malloc(sizef2);
	h_bs0 = (float *)malloc(sizef2);
	
	// host variable to keep all the possible errors
	h_errors = (float *)malloc(sizef);
	
	// host variable to hold the best error indexes, the lower one per block
	h_bei =(unsigned int *)malloc(sizeui);
	
	// host variable to hold the feature values
	h_g = (float *)malloc(sizef);
	
	// host variable to hold polarities
	h_p = (char *)malloc(sizec);
	
	// check available memory in device
	if(!flag)
	{
		long unsigned int free_byte, total_byte;
		hipMemGetInfo(&free_byte, &total_byte);
		printf("available memory in device: %lu\n", free_byte);
	}
	
	// compute number of blocks per grid needed
	blocksPerGrid = (N + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK; 
	
	if(!flag)
	{
		printf("blocks in the grid: %d, threads per block: %d\n", blocksPerGrid, MAX_THREADS_PER_BLOCK);
		flag = true;
	}
}

extern "C"
void closeParallelization()
{
	for(unsigned int i = 0; i < getFeaturesLength(); i++)
		checkCudaErrors(hipFree(d_f[i].rectangles));
		
	checkCudaErrors(hipFree(d_f));
	checkCudaErrors(hipFree(d_X));
	checkCudaErrors(hipFree(d_Y));
	checkCudaErrors(hipFree(d_i));
	checkCudaErrors(hipFree(d_g));
	checkCudaErrors(hipFree(d_w));
	checkCudaErrors(hipFree(d_s1));
	checkCudaErrors(hipFree(d_s0));
	checkCudaErrors(hipFree(d_e));
	checkCudaErrors(hipFree(d_bei));
	checkCudaErrors(hipFree(d_BufKey));
	checkCudaErrors(hipFree(d_BufVal));
	free(h_bs1);
	free(h_bs0);
	free(h_g);
	free(h_errors);
	free(h_bei);
	free(h_p);
}

extern "C"
void computeFeatureValuesParallel(unsigned int featureIndex, unsigned int iterationIndex)
{	
	fIndex = featureIndex;
	tIndex = iterationIndex;
	computeFeatureValuesKernel<<<blocksPerGrid, MAX_THREADS_PER_BLOCK>>>(d_X, d_g, d_i, N, D, fIndex);
	checkCudaErrors(hipDeviceSynchronize());

	// copy the result for indexes and feature values
	//checkCudaErrors(hipMemcpy(*g, d_g, sizef, hipMemcpyDeviceToHost));
	
	/*if(fIndex == FEATURE)
	{
		for(int i = 9000; i < 9500; i++)
		{
			printf("*****%d: %.12f\n", i, (*g)[i]);
		}
	}*/
}

extern "C"
void sortFeatureValuesParallel()
{
	// sort direction, if ascending 1, decreasing 0
	const unsigned int DIR = 1;
	
	initMergeSort();
	checkCudaErrors(hipDeviceSynchronize());
	mergeSort(d_g, d_i, d_BufKey, d_BufVal, d_g, d_i, N, DIR);
    checkCudaErrors(hipDeviceSynchronize());
	closeMergeSort();
}

extern "C"
void computeErrorParallel(float **w, float t1, float t0, float *finalError, 
					float *threshold, char *polarity)
{
	int sharedMemoryAmount = MAX_THREADS_PER_BLOCK * sizeof(float) * 2;
	
	// copy the updated weights to the device and perform scan per block
	checkCudaErrors(hipMemcpy(d_w, *w, sizef, hipMemcpyHostToDevice));
	computeWeightScanKernel<<<blocksPerGrid, MAX_THREADS_PER_BLOCK, sharedMemoryAmount>>>(d_Y, d_w, d_i, d_s1, d_s0, d_ss1, d_ss0, blocksPerGrid, fIndex);
	checkCudaErrors(hipDeviceSynchronize());
	
	// copy the last value of every scan block for s1 and s0
	checkCudaErrors(hipMemcpy(h_bs1, d_s1, sizef2, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_bs0, d_s0, sizef2, hipMemcpyDeviceToHost));
	
	// sum every last block value
	for(int b = 2; b < blocksPerGrid; b++)
	{
		h_bs1[b] = h_bs1[b] + h_bs1[b - 1];
		h_bs0[b] = h_bs0[b] + h_bs0[b - 1];
	}
	
	// copy back to the device
	checkCudaErrors(hipMemcpy(d_s1, h_bs1, sizef2, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_s0, h_bs0, sizef2, hipMemcpyHostToDevice));
	
	// compute error
	sharedMemoryAmount = (MAX_THREADS_PER_BLOCK * sizeof(float)) + (MAX_THREADS_PER_BLOCK * sizeof(int));
	computeErrorKernel<<<blocksPerGrid, MAX_THREADS_PER_BLOCK, sharedMemoryAmount>>>(d_e, d_bei, d_s1, d_s0, d_ss1, d_ss0, d_p, t1, t0, fIndex);
	checkCudaErrors(hipDeviceSynchronize());
	
	// copy feature values, errors, indexes and polarities to the host
	checkCudaErrors(hipMemcpy(h_g, d_g, sizef, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_errors, d_e, sizef, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_bei, d_bei, sizeui2, hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(h_p, d_p, sizec, hipMemcpyDeviceToHost));
	
	// get the lower error given by blocks and get the best error index
	*finalError = 7.7;
	unsigned int finalIndex = 0;

	for(int i = 0; i < blocksPerGrid; i++)
	{
		unsigned int idx = h_bei[i] + i * MAX_THREADS_PER_BLOCK;

		if(abs(h_errors[idx]) < *finalError)
		{
			*finalError = h_errors[idx];
			finalIndex = idx;
			*polarity = h_p[idx]; 
		}
	}
	
	*threshold = h_g[finalIndex + 1];
	
	/*if(fIndex == FEATURE)
	{
		float *ss1 = (float *)malloc(sizef);
		float *ss0 = (float *)malloc(sizef);
		unsigned int *ind = (unsigned int *)malloc(sizeui);
		float *gg = (float *)malloc(sizef);
		
		hipMemcpy(ss1, d_ss1, sizef, hipMemcpyDeviceToHost);
		hipMemcpy(ss0, d_ss0, sizef, hipMemcpyDeviceToHost);
		hipMemcpy(ind, d_i, sizeui, hipMemcpyDeviceToHost);
		hipMemcpy(gg, d_g, sizef, hipMemcpyDeviceToHost);
		
		for(int i = 5000; i < 5550; i++)
			printf("%d: %d, g: %.12f\n", i, ind[i], gg[i]);
	
		for(int i = LOWERID; i < UPPERID; i++)
			printf("(CPU) %d, s1:%.12f, s0:%.12f\n", i, ss1[i], ss0[i]);
			
		free(ss1); free(ss0); free(ind);
		//char c = getchar(); c++;
	}*/

	/*if(fIndex == FEATURE)
	{
		//printf("finalError: %.6f\n", *finalError);
		//printf("finalIndex: %d\n", *finalIndex);
		float ee = 7;
		unsigned int ei = 0;
		
		for(int i = 0; i < N; i++){
			if(errors[i] < ee){
				ee = errors[i];
				ei = i;
			}
		}
		
		printf("The truth -> bestIndex: %d (%d), bestError: %.12f (%.12f)\n", ei, *finalIndex, ee, *finalError);
		//char c = getchar(); c++;
	}*/
}